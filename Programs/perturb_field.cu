#include "hip/hip_runtime.h"
#define BLOCK_SIZE 8
#include <pycuda-complex.hpp>
#define INDEX(k,j,i,ld) ((k)*ld*ld + (j) * ld + (i))
#define E (float) (2.7182818284)

__global__ void set_velocity(pycuda::complex<float>* fourierbox, pycuda::complex<float>* vbox, 
float dDdt_overD, int w, int comp)
{
  int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
  int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
  int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
  int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
  int p = INDEX(k,j,i,w);
  if (j >= w || i >= w || k >= w) return;
  float k_x, k_y, k_z, k_sq;
  int hw = w/2; 
  k_z = (k>hw) ? (k-w)*%(DELTAK)s : k*%(DELTAK)s;
  k_y = (j>hw) ? (j-w)*%(DELTAK)s : j*%(DELTAK)s;
  k_x = (i>hw) ? (i-w)*%(DELTAK)s : i*%(DELTAK)s;

  k_sq = k_x*k_x + k_y*k_y + k_z*k_z;
  if (k_sq == 0)
  {
    vbox[p] = 0.0;
    return;
  }
  pycuda::complex<float> I = pycuda::complex<float>(0.f, 1.f);
  pycuda::complex<float> factor;
  switch (comp) {
    case 0:
      factor = k_x*dDdt_overD*I/k_sq;
    case 1:
      factor = k_y*dDdt_overD*I/k_sq;
    case 2:
      factor = k_z*dDdt_overD*I/k_sq;
  vbox[p] = factor * fourierbox[p];
  }
}


__global__ void move_mass(float* updated, float* deltax, float* vx, float* vy, float* vz, float init_growth_factor)
{
	int w = %(DIM)s;
	int sw = %(HII_DIM)s;

	int tx = threadIdx.x;  int ty = threadIdx.y; int tz = threadIdx.z;
	int bx = blockIdx.x;   int by = blockIdx.y; int bz = blockIdx.z;
	int bdx = blockDim.x;  int bdy = blockDim.y; int bdz = blockDim.z;
	int i = bdx * bx + tx; int j = bdy * by + ty; int k = bdz * bz + tz;
	int p = INDEX(k,j,i,w);
	float xf = (i+0.5)/w;
	float yf = (j+0.5)/w;
	float zf = (k+0.5)/w;
	int HII_i = floor(i/%(PIXEL_FACTOR)s);
	int HII_j = floor(j/%(PIXEL_FACTOR)s);
	int HII_k = floor(k/%(PIXEL_FACTOR)s);
	xf += vx[INDEX(HII_i, HII_j, HII_k, sw)];
	yf += vy[INDEX(HII_k, HII_j, HII_i, sw)];
	zf += vz[INDEX(HII_k, HII_j, HII_i, sw)];
	xf *= sw; yf *= sw; zf *= sw;

	while (xf >= (float)sw){ xf -= sw;}
	while (xf < 0){ xf += sw;}
	while (yf >= (float)sw){ yf -= sw;}
	while (yf < 0){ yf += sw;}
	while (zf >= (float)sw){ zf -= sw;}
	while (zf < 0){ zf += sw;}
	int xi = xf; 
	int yi = yf; 
	int zi = zf;
	if (xi >= sw){ xi -= sw;}
	if (xi < 0) {xi += sw;}
	if (yi >= sw){ yi -= sw;}
	if (yi < 0) {yi += sw;}
	if (zi >= sw){ zi -= sw;}
	if (zi < 0) {zi += sw;}

	// move the mass
	updated[INDEX(zi, yi, xi, sw)] += (1 + init_growth_factor*deltax[p]);

	}